#include <string.h>
#include <time.h>
//#include <sys/time.h>
#include <typeinfo>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <hipblas.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stdint.h>
#include "matrix.h"



void test_gpu1()
{
    int n = 256 ;
    char *a = (char *)calloc(n * n, sizeof(char));
    char *b = (char *)calloc(n * n, sizeof(char));
    for (int i = 0; i < n ; i ++){
        for (int j = 0; j < n; j ++)
        {
            a[i * n + j] = (char)(1 + i);
            b[i * n + j] = (char)(1 + j);
        }
    }
    int *c = (int*)calloc(n * n, sizeof(int));
    cucblas_ggemm(a, b ,c);
    for (int i= 0; i < 4; i ++){
        printf("%f \n", (float)c[i]);
    }
}

void test_gpu2()
{
    int n = 256;
    float *a = (float *)calloc(n * n, sizeof(float));
    float *b = (float *)calloc(n * n, sizeof(float));
    float *c = (float *)calloc(n * n, sizeof(float));
    for (int i = 0; i < n; i ++){
        for (int j = 0; j < n; j ++)
        {
            a[i * n + j] = (1 + i);
            b[i * n + j] = (1 + j);
        }
    }
    cucblas_sgemm(a, b ,c);
    for (int i= 0; i < 4; i ++){
        printf("%f \n", c[i]);
    }
}

void cucblas_ggemm(char *h_ma, char *h_mb, int *h_mc) {


    int n = 256;
    int k = 256;
    int m = 256;
    char *mat1_g;
    char *mat2_g;
    int *mat3_g;
    int alpha = 1;
    int beta = 0;
    
    int devID = 0;
    hipSetDevice(devID);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, devID);
    printf("Device : %s, compute SM %d.%d.\n",devProp.name, devProp.major, devProp.minor);
    hipMalloc((void **)&(mat1_g), sizeof(char) * m * k);
    hipMalloc((void **)&(mat2_g), sizeof(char) * n * k);
    hipMalloc((void **)&(mat3_g), sizeof(int) * m * n);
    
    hipMemcpy(mat1_g, h_ma, sizeof(char) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(mat2_g, h_mb, sizeof(char) * n * n, hipMemcpyHostToDevice);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, m, k, &alpha, mat1_g, HIP_R_8I, n, mat2_g, HIP_R_8I, n, &beta, mat3_g, HIP_R_32I, n, HIP_R_32I, HIPBLAS_GEMM_DEFAULT);
    hipblasDestroy(handle);
    hipDeviceSynchronize();
    hipMemcpy(h_mc, mat3_g, sizeof(int) * n * n, hipMemcpyDeviceToHost);
    hipFree(mat1_g);
    hipFree(mat2_g);
    hipFree(mat3_g);
}

void cucblas_sgemm(float *h_ma, float *h_mb, float *h_mc) {


    int n = 256;
    int k = 256;
    int m = 256;
    hipblasOperation_t opt1 = HIPBLAS_OP_T;
    hipblasOperation_t opt2 = HIPBLAS_OP_T;

    float *mat1_g;
    float *mat2_g;
    float *mat3_g;
    float alpha = 1.0;
    float beta = 0.0;
    
    hipMalloc((void **)&(mat1_g), sizeof(float) * m * k);
    hipMalloc((void **)&(mat2_g), sizeof(float) * n * k);
    hipMalloc((void **)&(mat3_g), sizeof(float) * m * n);
    
    hipMemcpy(mat1_g, h_ma, sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(mat2_g, h_mb, sizeof(float) * n * n, hipMemcpyHostToDevice);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    hipblasSgemm(handle, opt1, opt2, m, n, k, &alpha, mat1_g, n, mat2_g, n, &beta, mat3_g, n);
    //hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, m, k, &alpha, mat1_g, HIP_R_16F, n, mat2_g, HIP_R_16F, n, &beta, mat3_g, HIP_R_32F, n, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    hipblasDestroy(handle);
    hipDeviceSynchronize();
    hipMemcpy(h_mc, mat3_g, sizeof(float) * n * n, hipMemcpyDeviceToHost);
    hipFree(mat1_g);
    hipFree(mat2_g);
    hipFree(mat3_g);
}
